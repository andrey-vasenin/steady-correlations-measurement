#include "hip/hip_runtime.h"
﻿//
// Created by andrei on 4/13/21.
//
#include <memory>
#include <iostream>
#include <functional>
#include <vector>
#include <numeric>
#include <complex>
#include <cstdint>
#include "digitizer.h"
#include "dsp.cuh"
#include "measurement.cuh"
#include <pybind11/pybind11.h>

namespace py = pybind11;

Measurement::Measurement(std::uintptr_t dig_handle, unsigned long long averages, int batch, float part)
{
    dig = new Digitizer(reinterpret_cast<void*>(dig_handle));
    segment_size = dig->getSegmentSize();
    batch_size = batch;
    this->setAveragesNumber(averages);
    notify_size = 2 * segment_size * batch_size;
    dig->handleError();
    dig->setTimeout(5000);  // ms
    processor = new dsp(segment_size, batch_size, part);
    this->initializeBuffer();

    func = [this](const char* data) mutable { this->processor->compute(data); };

    int trace_length = processor->getTraceLength();

    test_input = new char[notify_size * 2];
}
void Measurement::initializeBuffer()
{
    // Create the buffer in page-locked memory
    int buffersize = 4 * notify_size;
    processor->createBuffer(buffersize * sizeof(char));
    auto buffer = processor->getBufferPointer();
    dig->setBuffer(buffer, buffersize);
}

void Measurement::setAmplitude(int ampl)
{
    processor->setAmplitude(ampl);
}

/* Use frequency in GHz */
void Measurement::setIntermediateFrequency(float frequency)
{
    int oversampling = (int)std::round(1.25E+9f / dig->getSamplingRate());
    processor->setIntermediateFrequency(frequency, oversampling);
    hipDeviceSynchronize();
}

void Measurement::setAveragesNumber(unsigned long long averages)
{
    segments_count = averages;
    iters_num = static_cast<int>(averages / static_cast<unsigned long long>(batch_size));
    iters_done = 0;
}

void Measurement::setCalibration(float r, float phi, float offset_i, float offset_q)
{
    processor->setDownConversionCalibrationParameters(r, phi, offset_i, offset_q);
}

void Measurement::setFirwin(float left_cutoff, float right_cutoff)
{
    int oversampling = (int)std::round(1.25E+9f / dig->getSamplingRate());
    processor->setFirwin(left_cutoff, right_cutoff, oversampling);
    hipDeviceSynchronize();
}

int Measurement::getCounter()
{
    return processor->getCounter();
}

void Measurement::measure()
{
    dig->launchFifo(notify_size, iters_num, func);
    iters_done += iters_num;
}

void Measurement::measureTest()
{
    for (int i = 0; i < iters_num; i++)
        func(test_input);
    iters_done += iters_num;
}

std::vector<double> Measurement::getPSD()
{

}

void Measurement::setTestInput(py::buffer input)
{
    py::buffer_info info = input.request();
    if (info.ndim != 1)
        throw std::runtime_error("Number of dimensions must be one");
    if (info.size < 2 * segment_size)
        throw std::runtime_error("Number of element in the imput array "
            "must be larger or equal to the two segment sizes");

    char* input_ptr = (char*)info.ptr;

    for (int j = 0; j < batch_size; j++)
    {
        for (int i = 0; i < segment_size; i++)
        {
            test_input[2 * (j * segment_size + i)] = (char)input_ptr[2 * i];
            test_input[2 * (j * segment_size + i) + 1] = (char)input_ptr[2 * i + 1];
        }
    }
}

void Measurement::setSubtractionTrace(std::vector<std::complex<float>> trace)
{
    //using namespace std::complex_literals;
    int N = processor->getTraceLength();
    int M = processor->getTotalLength();

    //py::buffer_info info = trace.request();
    //if (info.ndim != 1)
    //    throw std::runtime_error("Number of dimensions must be one");
    //if (info.size < 2 * N)
    //    throw std::runtime_error("Number of element in the input array "
    //        "must be larger or equal to the returned trace lenght");

    //float* trace_ptr = (float*)info.ptr;

    std::vector<std::complex<float>> average(M);

    for (int i = 0; i < batch_size; i++)
    {
        for (int j = 0; j < N; j++)
        {
            //average[i * N + j] = std::complex<float>(trace_ptr[2 * j], trace_ptr[2 * j + 1]);
            average[i * N + j] = trace[j];
        }
    }

    processor->setSubtractionTrace(average);
}

std::vector<std::complex<float>> Measurement::getSubtractionTrace()
{
    int len = processor->getTotalLength();
    std::vector<std::complex<float>> subtraction_trace(len);
    processor->getSubtractionTrace(subtraction_trace);
    return subtraction_trace;
}

void Measurement::reset()
{
    this->resetOutput();
    processor->resetSubtractionTrace();
}

void Measurement::resetOutput()
{
    iters_done = 0;
    processor->resetOutput();
}

void Measurement::free()
{
    delete processor;
    delete dig;
    processor = NULL;
    dig = NULL;
    delete[] test_input;
}

Measurement::~Measurement()
{
    if ((processor != NULL) || (dig != NULL))
        this->free();
}